#include "hip/hip_runtime.h"
#pragma comment(lib, "cufft.lib")

#include <iostream>
#include <vector>
#include <complex>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <fstream>
#include <string>

typedef std::complex<double> Complex;

__global__ void cuFFT(std::vector<Complex>& input) {
	
	// initialize CUDA
	hipSetDevice(0);

	// create cuFFT handle
	hipfftHandle plan;
	hipfftPlan1d(&plan, input.size(), HIPFFT_Z2Z, 1);

	// host to device
	Complex* d_data;
	hipMalloc((void**)&d_data, sizeof(Complex) * input.size());
    hipMemcpy(d_data, input.data(), sizeof(Complex) * input.size(), hipMemcpyHostToDevice);

    // run FFT
    hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_data, (hipfftDoubleComplex*)d_data, HIPFFT_FORWARD);

    // device to host
    std::vector<Complex> result(input.size());
    hipMemcpy(result.data(), d_data, sizeof(Complex) * input.size(), hipMemcpyDeviceToHost);

    // print the result
    std::cout << "Result of cuFFT:" << std::endl;
    for (size_t i = 0; i < result.size(); ++i) {
        std::cout << result[i] << std::endl;
    }

    // Free memory
    hipFree(d_data);
    hipfftDestroy(plan);
}

// read from file
std::vector<Complex> readDataFromFile(const std::string& filename) {
    std::ifstream inputFile(filename, std::ios::binary);
    if (!inputFile.is_open()) {
        std::cerr << "Cannot open the file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    // Calculate the file size
    inputFile.seekg(0, std::ios::end);
    size_t fileSize = inputFile.tellg();
    inputFile.seekg(0, std::ios::beg);

    // Make file data
    std::vector<Complex> data(fileSize / sizeof(Complex));

    // read data from file
    inputFile.read(reinterpret_cast<char*>(data.data()), fileSize);
    inputFile.close();

    return data;
}

__global__ void file_cuFFT(std::string filename) {

    std::vector<Complex> input = readDataFromFile(filename);

    // initialize CUDA
    hipSetDevice(0);

    // create cuFFT handle
    hipfftHandle plan;
    hipfftPlan1d(&plan, input.size(), HIPFFT_Z2Z, 1);

    // host to device
    Complex* d_data;
    hipMalloc((void**)&d_data, sizeof(Complex) * input.size());
    hipMemcpy(d_data, input.data(), sizeof(Complex) * input.size(), hipMemcpyHostToDevice);

    // run FFT
    hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_data, (hipfftDoubleComplex*)d_data, HIPFFT_FORWARD);

    // device to host
    std::vector<Complex> result(input.size());
    hipMemcpy(result.data(), d_data, sizeof(Complex) * input.size(), hipMemcpyDeviceToHost);

    // print the result
    std::cout << "Result of cuFFT:" << std::endl;
    for (size_t i = 0; i < result.size(); ++i) {
        std::cout << result[i] << std::endl;
    }

    // Free memory
    hipFree(d_data);
    hipfftDestroy(plan);
}