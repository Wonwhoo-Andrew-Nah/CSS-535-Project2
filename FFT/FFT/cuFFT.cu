// compile with following prompt
// $ nvcc -o output example.cu -lcufft

#include <stdio.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define PI 3.14159265358979323846
#define CUDA_CHECK(call) checkCudaError(call, __FILE__, __LINE__)

void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << "at" << file << ":" << line << std::endl;
        exit(-1);
    }
}

int main() {
    
    int N = 16;

    // Host memory
    hipDoubleComplex* h_data = new hipDoubleComplex[N];
    for (int i = 0; i < N; ++i) {
        h_data[i].x = i; // real
        h_data[i].y = 0; // imaginary
    }

    // Copy to Device
    hipDoubleComplex* d_data;
    CUDA_CHECK(hipMalloc((void**)&d_data, N * sizeof(hipDoubleComplex)));
    CUDA_CHECK(hipMemcpy(d_data, h_data, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // Instantiate cuFFT plan
    hipfftHandle plan;
    hipfftResult_t result = hipfftPlan1d(&plan, N, HIPFFT_Z2Z, 1);
    if (result != HIPFFT_SUCCESS) {
        std::cerr << "cuFFT plan creation failed" << std::endl;
        exit(-1);
    }

    // run cuFFT
    result = hipfftExecZ2Z(plan, d_data, d_data, HIPFFT_FORWARD); // Run Forward FFT
    if (result != HIPFFT_SUCCESS) {
        std::cerr << "cuFFT execution failed" << std::endl;
        exit(-1);
    }

    // print the results
    CUDA_CHECK(hipMemcpy(h_data, d_data, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    std::cout << "cuFFT :" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << "(" << h_data[i].x << ", " << h_data[i].y << ")" << std::endl;
    }

    // free memory
    hipfftDestroy(plan);
    CUDA_CHECK(hipFree(d_data));
    delete[] h_data;

    return 0;

    //unoptimized
    //(1, 0)
    //(3.92388, -0.382683)
    //(7.55487, -1.47247)
    //(11.5685, -3.48614)
    //(15.5822, -6.49981)
    //(19.2132, -10.4374)
    //(22.1371, -15.082)
    //(24.1371, -20.1094)
    //(8.13707, -25.1367)
    //(2.89828, -26.3372)
    //(-0.325033, -25.0588)
    //(-0.737654, -22.2616)
    //(2.02771, -19.1367)
    //(7.85455, -16.9252)
    //(16.1602, -16.7293)
    //(25.9848, -19.344)

    //Z2Z
    //(120, 0)
    //(-8, 40.2187)
    //(-8, 19.3137)
    //(-8, 11.9728)
    //(-8, 8)
    //(-8, 5.34543)
    //(-8, 3.31371)
    //(-8, 1.5913)
    //(-8, 0)
    //(-8, -1.5913)
    //(-8, -3.31371)
    //(-8, -5.34543)
    //(-8, -8)
    //(-8, -11.9728)
    //(-8, -19.3137)
    //(-8, -40.2187)
}
