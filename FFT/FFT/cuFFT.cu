#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <complex>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

typedef std::complex<double> Complex;

__global__ void cuFFT(std::vector<Complex>& input) {
	
	// initialize CUDA
	hipSetDevice(0);

	// create cuFFT handle
	hipfftHandle plan;
	hipfftPlan1d(&plan, input.size(), HIPFFT_Z2Z, 1);

	// host to device
	Complex* d_data;
	hipMalloc((void**)&d_data, sizeof(Complex) * input.size());
    hipMemcpy(d_data, input.data(), sizeof(Complex) * input.size(), hipMemcpyHostToDevice);

    // run FFT
    hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_data, (hipfftDoubleComplex*)d_data, HIPFFT_FORWARD);

    // device to host
    std::vector<Complex> result(input.size());
    hipMemcpy(result.data(), d_data, sizeof(Complex) * input.size(), hipMemcpyDeviceToHost);

    // print the result
    std::cout << "Result of cuFFT:" << std::endl;
    for (size_t i = 0; i < result.size(); ++i) {
        std::cout << result[i] << std::endl;
    }

    // Free memory
    hipFree(d_data);
    hipfftDestroy(plan);
}