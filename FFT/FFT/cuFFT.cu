// compile with following prompt
// $ nvcc -o output example.cu -lcufft

#include <stdio.h>
#include <vector>
#include <math.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define PI 3.14159265358979323846
#define CUDA_CHECK(call) checkCudaError(call, __FILE__, __LINE__)

void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << "at" << file << ":" << line << std::endl;
        exit(-1);
    }
}

int main() {
    
    std::vector<int> input_sizes = { 2<<1, 2<<3, 2<<4, 2<<6, 2<<8, 2<<16, 2<<20 };

    for (int N : input_sizes) {
        // Host memory
        hipDoubleComplex* h_data = new hipDoubleComplex[N];
        for (int i = 0; i < N; ++i) {
            h_data[i].x = sin(2 * PI * i / N); // real
            h_data[i].y = 0; // imaginary
        }

        // Copy to Device
        hipDoubleComplex* d_data;
        CUDA_CHECK(hipMalloc((void**)&d_data, N * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemcpy(d_data, h_data, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        // Instantiate cuFFT plan
        hipfftHandle plan;
        hipfftResult_t result = hipfftPlan1d(&plan, N, HIPFFT_Z2Z, 1);
        if (result != HIPFFT_SUCCESS) {
            std::cerr << "cuFFT plan creation failed" << std::endl;
            exit(-1);
        }

        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        // Start timing
        hipEventRecord(start);

        // Run cuFFT
        result = hipfftExecZ2Z(plan, d_data, d_data, HIPFFT_FORWARD);
        if (result != HIPFFT_SUCCESS) {
            std::cerr << "cuFFT execution failed" << std::endl;
            exit(-1);
        }

        // End timing
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // Calculate elapsed time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Print the results
        std::cout << "Input size: " << N << std::endl;
        std::cout << "cuFFT execution time: " << milliseconds << " ms" << std::endl;

        // Cleanup
        hipfftDestroy(plan);
        CUDA_CHECK(hipFree(d_data));
        delete[] h_data;

        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    }

    return 0;
}
