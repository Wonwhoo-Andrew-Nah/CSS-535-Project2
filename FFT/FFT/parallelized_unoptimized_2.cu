#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <vector>
#include <hip/hip_complex.h>
#include <chrono>

#define PI 3.14159265358979323846
#define CUDA_CHECK(call) checkCudaError(call, __FILE__, __LINE__)

void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << "at" << file << ":" << line << std::endl;
        exit(-1);
    }
}

__device__ hipDoubleComplex w(int N, int k) {
    return make_hipDoubleComplex(cos(-2 * PI * k / N), sin(-2 * PI * k / N));
}

__global__ void fftKernel(hipDoubleComplex *data, int N, int offset) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N / 2) {
        int k = idx + offset;
        hipDoubleComplex W = w(N, k);
        hipDoubleComplex even = data[2 * idx];
        hipDoubleComplex odd = hipCmul(W, data[2 * idx + 1]);
        data[idx] = hipCadd(even, odd);
        data[idx + N / 2] = hipCsub(even, odd);
    }
}

int main() {

    std::vector<int> input_sizes = { 2<<1, 2<<3, 2<<4, 2<<6, 2<<8, 2<<16, 2<<20 };

    for (int N : input_sizes) {
        
        hipDoubleComplex* h_signal = new hipDoubleComplex[N];
        for (int i = 0; i < N; ++i) {
            // h_data[i].x = sin(2 * PI * i / N); // real
            h_signal[i].x = i; // real
            h_signal[i].y = 0; // imaginary
        }

        hipDoubleComplex* d_signal;
        CUDA_CHECK(hipMalloc((void**)&d_signal, N * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemcpy(d_signal, h_signal, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        CUDA_CHECK(hipEventRecord(start));
        for (int size = 2; size <= N; size <<= 1) {
            int numBlocks = (N / size) / 256 + 1;
            fftKernel<<<numBlocks, 256>>>(d_signal, N, N / size);
        }   
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

        std::cout << "Input size: " << N << std::endl;
        std::cout << "Execution time: " << milliseconds << " ms" << std::endl;

        hipDoubleComplex *h_result = new hipDoubleComplex[N];
        CUDA_CHECK(hipMemcpy(h_result, d_signal, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

        // print first 16 results
        std::cout << "First 4 elements of the result:" << std::endl;
        for (int i = 0; i < 4; ++i) {
            std::cout << "(" << h_result[i].x << ", " << h_result[i].y << ") " << std::endl;
        }
        std::cout << std::endl;

        delete[] h_signal;
        delete[] h_result;
        CUDA_CHECK(hipFree(d_signal));
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    }

    return 0;
}
