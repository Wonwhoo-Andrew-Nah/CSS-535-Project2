
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_complex.h>

#define PI 3.14159265358979323846

__global__ void direct_dft(hipDoubleComplex *in, hipDoubleComplex *out, int n)
{
        int threadID = blockIdx.x * blockDim.x + threadIdx.x;

        if (threadID < n)
        {
                hipDoubleComplex sum = make_hipDoubleComplex(0, 0);

                for (int i = 0; i < n; i++)
                {
                        double angle = -2.0 * PI * threadID * i / n;

                        hipDoubleComplex tSub1 
                                = make_hipDoubleComplex(cos(angle), 0);

                        hipDoubleComplex tSub2 
                                = hipCmul(make_hipDoubleComplex(0, 1), 
                                         make_hipDoubleComplex(sin(angle), 0));

                        hipDoubleComplex t1 = hipCmul(in[i], tSub1);
                        hipDoubleComplex t2 = hipCmul(in[i], tSub2);

                        hipDoubleComplex t = hipCadd(t1, t2);

                        sum = hipCadd(sum, t);
                }

                out[threadID] = sum;
        }
}

int main()
{
        int n = 16;

        hipDoubleComplex *h_in = (hipDoubleComplex *)malloc(n * sizeof(hipDoubleComplex));
        hipDoubleComplex *h_out = (hipDoubleComplex *)malloc(n * sizeof(hipDoubleComplex));

        hipDoubleComplex *d_in;
        hipMalloc(&d_in, n * sizeof(hipDoubleComplex));

        hipDoubleComplex *d_out;
        hipMalloc(&d_out, n * sizeof(hipDoubleComplex));


        printf("in:\n");
        h_in[0] = make_hipDoubleComplex((double) 0, 0);
        h_in[1] = make_hipDoubleComplex((double) 1, 0);
        h_in[2] = make_hipDoubleComplex((double) 2, 0);
        h_in[3] = make_hipDoubleComplex((double) 3, 0);
        h_in[4] = make_hipDoubleComplex((double) 4, 0);
        h_in[5] = make_hipDoubleComplex((double) 5, 0);
        h_in[6] = make_hipDoubleComplex((double) 6, 0);
        h_in[7] = make_hipDoubleComplex((double) 7, 0);
        h_in[8] = make_hipDoubleComplex((double) 8, 0);
        h_in[9] = make_hipDoubleComplex((double) 9, 0);
        h_in[10] = make_hipDoubleComplex((double) 10, 0);
        h_in[11] = make_hipDoubleComplex((double) 11, 0);
        h_in[12] = make_hipDoubleComplex((double) 12, 0);
        h_in[13] = make_hipDoubleComplex((double) 13, 0);
        h_in[14] = make_hipDoubleComplex((double) 14, 0);
        h_in[15] = make_hipDoubleComplex((double) 15, 0);

        for (int i = 0; i < n; i++)
        {
                printf("%f, %f\n", hipCreal(h_in[i]), hipCimag(h_in[i]));
        }

        hipMemcpy(d_in, h_in, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

        direct_dft<<<4, 4>>>(d_in, d_out, n);

        hipMemcpy(h_out, d_out, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

        printf("\nout:\n");
        for (int i = 0; i < n; i++)
        {
                printf("%f, %f\n", hipCreal(h_out[i]), hipCimag(h_out[i]));
        }

        free(h_in);
        free(h_out);
        hipFree(d_in);
        hipFree(d_out);

        return 0;
}
