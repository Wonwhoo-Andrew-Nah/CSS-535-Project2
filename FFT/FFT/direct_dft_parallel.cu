#include <stdio.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define PI 3.14159265358979323846
#define CUDA_CHECK(call) checkCudaError(call, __FILE__, __LINE__)
void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << "at" << file << ":" << line << std::endl;
        exit(-1);
    }
}

__global__ void direct_dft(hipDoubleComplex *in, hipDoubleComplex *out, int n)
{
        int threadID = blockIdx.x * blockDim.x + threadIdx.x;

        if (threadID < n)
        {
                hipDoubleComplex sum = make_hipDoubleComplex(0, 0);

                for (int i = 0; i < n; i++)
                {
                        double angle = -2.0 * PI * threadID * i / n;

                        hipDoubleComplex tSub1 
                                = make_hipDoubleComplex(cos(angle), 0);

                        hipDoubleComplex tSub2 
                                = hipCmul(make_hipDoubleComplex(0, 1), 
                                         make_hipDoubleComplex(sin(angle), 0));

                        hipDoubleComplex t1 = hipCmul(in[i], tSub1);
                        hipDoubleComplex t2 = hipCmul(in[i], tSub2);

                        hipDoubleComplex t = hipCadd(t1, t2);

                        sum = hipCadd(sum, t);
                }

                out[threadID] = sum;
        }
}

int main()
{
        // int n = 16;
        std::vector<int> input_sizes = { 2<<1, 2<<3, 2<<4, 2<<6, 2<<8, 2<<16, 2<<20 };

        for (int n : input_sizes) {
        
                hipDoubleComplex *h_in = (hipDoubleComplex *)malloc(n * sizeof(hipDoubleComplex));
                hipDoubleComplex *h_out = (hipDoubleComplex *)malloc(n * sizeof(hipDoubleComplex));

                hipDoubleComplex *d_in;
                hipMalloc(&d_in, n * sizeof(hipDoubleComplex));

                hipDoubleComplex *d_out;
                hipMalloc(&d_out, n * sizeof(hipDoubleComplex));

                for (int i = 0; i < n; ++i) {
                        // h_data[i].x = sin(2 * PI * i / N); // real
                        h_in[i].x = (double) i; // real
                        h_in[i].y = 0; // imaginary
                }

                // printf("in:\n");
                // h_in[0] = make_cuDoubleComplex((double) 0, 0);
                // h_in[1] = make_cuDoubleComplex((double) 1, 0);
                // h_in[2] = make_cuDoubleComplex((double) 2, 0);
                // h_in[3] = make_cuDoubleComplex((double) 3, 0);
                // h_in[4] = make_cuDoubleComplex((double) 4, 0);
                // h_in[5] = make_cuDoubleComplex((double) 5, 0);
                // h_in[6] = make_cuDoubleComplex((double) 6, 0);
                // h_in[7] = make_cuDoubleComplex((double) 7, 0);
                // h_in[8] = make_cuDoubleComplex((double) 8, 0);
                // h_in[9] = make_cuDoubleComplex((double) 9, 0);
                // h_in[10] = make_cuDoubleComplex((double) 10, 0);
                // h_in[11] = make_cuDoubleComplex((double) 11, 0);
                // h_in[12] = make_cuDoubleComplex((double) 12, 0);
                // h_in[13] = make_cuDoubleComplex((double) 13, 0);
                // h_in[14] = make_cuDoubleComplex((double) 14, 0);
                // h_in[15] = make_cuDoubleComplex((double) 15, 0);

                // for (int i = 0; i < n; i++)
                // {
                //         printf("%f, %f\n", cuCreal(h_in[i]), cuCimag(h_in[i]));
                // }

                hipMemcpy(d_in, h_in, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

                hipEvent_t start, stop;
                CUDA_CHECK(hipEventCreate(&start));
                CUDA_CHECK(hipEventCreate(&stop));

                CUDA_CHECK(hipEventRecord(start));

                direct_dft<<<4, 4>>>(d_in, d_out, n);
                CUDA_CHECK(hipEventRecord(stop));
                CUDA_CHECK(hipEventSynchronize(stop));
                float milliseconds = 0;
                CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

                std::cout << "Input size: " << n << std::endl;
                std::cout << "Execution time: " << milliseconds << " ms" << std::endl;
                hipMemcpy(h_out, d_out, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

                // printf("\nout:\n");
                // for (int i = 0; i < n; i++)
                // {
                //         printf("%f, %f\n", cuCreal(h_out[i]), cuCimag(h_out[i]));
                // }

                free(h_in);
                free(h_out);
                hipFree(d_in);
                hipFree(d_out);
                CUDA_CHECK(hipEventDestroy(start));
                CUDA_CHECK(hipEventDestroy(stop));
        }
        return 0;
}
