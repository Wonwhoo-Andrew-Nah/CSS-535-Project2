#include "hip/hip_runtime.h"
#include "parallelized_unoptimized.h"

#define PI 3.14159265358979323846

__device__ void fft(hipDoubleComplex* signal, int n, int offset, int step) {
	if (n <= 1) return;

	hipDoubleComplex even = signal[offset];
	hipDoubleComplex odd = signal[offset + step];

	fft(signal, n / 2, offset, step * 2);
	fft(signal, n / 2, offset + step, step * 2);

	for (int i = 0; i < n / 2; i++)
	{
		double angle = -2 * PI * i / n;

		hipDoubleComplex temp;
		temp.x = cos(angle) * hipCreal(odd) - sin(angle) * hipCimag(odd);
		temp.y = cos(angle) * hipCimag(odd) + sin(angle) * hipCreal(odd);

		signal[offset + i] = hipCadd(even, temp);
		signal[offset + i + n / 2] = hipCsub(even, temp);

		even = signal[offset + i + step];
	}
}

__global__ void perform_fft(hipDoubleComplex* signal, int n) {
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	fft(signal, n, threadID, 1);
}

void test_our_fft() {
	const int N = 40;
	hipDoubleComplex* signal;
	hipDoubleComplex* d_signal;
	signal = (hipDoubleComplex*)malloc(N * sizeof(hipDoubleComplex));
	for (int i = 0; i < N; i++) {
		signal[i] = make_hipDoubleComplex(sin(2 * PI * i / N), cos(2 * PI * i / N));
	}
	hipMalloc(&d_signal, N * sizeof(hipDoubleComplex));
	hipMemcpy(d_signal, signal, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	perform_fft << <1, N >> > (d_signal, N);
	hipMemcpy(signal, d_signal, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	printf("FFT Results:\n");
	for (int i = 0; i < N; i++) {
		printf("(%f, %f)\n", hipCreal(signal[i]), hipCimag(signal[i]));
	}
	hipFree(d_signal);
	free(signal);
}
