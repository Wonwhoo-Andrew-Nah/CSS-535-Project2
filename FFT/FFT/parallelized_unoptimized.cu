// $ nvcc -o output example.cu -lcufft

#include <stdio.h>
#include <vector>
#include <math.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define PI 3.14159265358979323846
#define CUDA_CHECK(call) checkCudaError(call, __FILE__, __LINE__)

__device__ void fft(hipDoubleComplex* signal, int n, int offset, int step) {
	if (n <= 1) return;

	hipDoubleComplex even = signal[offset];
	hipDoubleComplex odd = signal[offset + step];

	fft(signal, n / 2, offset, step * 2);
	fft(signal, n / 2, offset + step, step * 2);

	for (int i = 0; i < n / 2; i++) {
		double angle = -2 * PI * i / n;

		hipDoubleComplex temp;
		temp.x = cos(angle) * hipCreal(odd) - sin(angle) * hipCimag(odd);
		temp.y = cos(angle) * hipCimag(odd) + sin(angle) * hipCreal(odd);

		signal[offset + i] = hipCadd(even, temp);
		signal[offset + i + n / 2] = hipCsub(even, temp);

		even = signal[offset + i + step];
	}
}

__global__ void perform_fft(hipDoubleComplex* signal, int n) {
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	fft(signal, n, threadID, 1);
}

void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << "at" << file << ":" << line << std::endl;
        exit(-1);
    }
}

int main() {

	CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, 4 * 1024));

    std::vector<int> input_sizes = { 2<<1, 2<<3, 2<<4, 2<<6, 2<<8, 2<<16, 2<<20 };
    const int blockSize = 256;

    for (int N : input_sizes) {

        hipDoubleComplex* h_signal = new hipDoubleComplex[N];
        for (int i = 0; i < N; ++i) {
            h_signal[i].x = sin(2 * PI * i / N); // real
            h_signal[i].y = 0; // imaginary
        }

        hipDoubleComplex* d_signal;
        CUDA_CHECK(hipMalloc((void**)&d_signal, N * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemcpy(d_signal, h_signal, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        CUDA_CHECK(hipEventRecord(start));
        perform_fft <<<(N + blockSize - 1) / blockSize, blockSize>>> (d_signal, N);
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

        std::cout << "Input size: " << N << std::endl;
        std::cout << "Execution time: " << milliseconds << " ms" << std::endl;

        hipDoubleComplex *h_result = new hipDoubleComplex[N];
        CUDA_CHECK(hipMemcpy(h_result, d_signal, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        // for (int i = 0; i< 4; ++i) {
        //     std::cout << "(" << h_result[i].x << ", " << h_result[i].y << ")" << std::endl;
        // }

        delete[] h_result;
        delete[] h_signal;
        CUDA_CHECK(hipFree(d_signal));
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    }

    return 0;
}
