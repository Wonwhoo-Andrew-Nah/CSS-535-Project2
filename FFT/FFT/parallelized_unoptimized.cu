#include "hip/hip_runtime.h"
// $ nvcc -o output example.cu -lcufft

#include <stdio.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define PI 3.14159265358979323846
#define CUDA_CHECK(call) checkCudaError(call, __FILE__, __LINE__)

__device__ void fft(hipDoubleComplex *signal, int n, int offset, int step)
{
        if (n <= 1) return;

        hipDoubleComplex even = signal[offset];
        hipDoubleComplex odd = signal[offset + step];

        fft(signal, n / 2, offset, step * 2);
        fft(signal, n / 2, offset + step, step * 2);

        for (int i = 0; i < n / 2; i++)
        {
                double angle = -2 * PI * i / n;

                hipDoubleComplex temp;
                temp.x = cos(angle) * hipCreal(odd) - sin(angle) * hipCimag(odd);
                temp.y = cos(angle) * hipCimag(odd) + sin(angle) * hipCreal(odd);

                signal[offset + i] = hipCadd(even, temp);
                signal[offset + i + n / 2] = hipCsub(even, temp);

                even = signal[offset + i + step];
        }
}

__global__ void perform_fft(hipDoubleComplex *signal, int n)
{
        int threadID = blockIdx.x * blockDim.x + threadIdx.x;

        fft(signal, n, threadID, 1);
}

void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << "at" << file << ":" << line << std::endl;
        exit(-1);
    }
}

int main()
{       
        // length of the signal
        const int N = 16;
        const int blockSize = 8;

        // Host memory
        hipDoubleComplex *h_signal = new hipDoubleComplex[N];

        // input data initialization
        for (int i = 0; i < N; ++i){
                // real
                h_signal[i].x = i;
                // imaginary
                h_signal[i].y = 0;
        }

        // Copy to device
        hipDoubleComplex *d_signal;
        CUDA_CHECK(hipMalloc((void**)&d_signal, N * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemcpy(d_signal, h_signal, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        
        // call kernel function
        perform_fft << < (N + blockSize - 1) / blockSize, blockSize >> > (d_signal, N);
        
        // Copy to host
        hipDoubleComplex *h_result = new hipDoubleComplex[N];
        CUDA_CHECK(hipMemcpy(h_result, d_signal, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

        // check the results
        std::cout << "unoptimized" << std::endl;
        for (int i = 0; i< N; ++i){
                std::cout<< "(" << h_result[i].x << ", " << h_result[i].y << ")" << std::endl;
        }

        // deallocate memory
        delete[] h_signal;
        delete[] h_result;
        CUDA_CHECK(hipFree(d_signal));

        return 0;


        // unoptimized
        //(1, 0)
        //(3.92388, -0.382683)
        //(7.55487, -1.47247)
        //(11.5685, -3.48614)
        //(15.5822, -6.49981)
        //(19.2132, -10.4374)
        //(22.1371, -15.082)
        //(24.1371, -20.1094)
        //(8.13707, -25.1367)
        //(2.89828, -26.3372)
        //(-0.325033, -25.0588)
        //(-0.737654, -22.2616)
        //(2.02771, -19.1367)
        //(7.85455, -16.9252)
        //(16.1602, -16.7293)
        //(25.9848, -19.344)
}
