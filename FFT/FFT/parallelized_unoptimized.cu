#include "hip/hip_runtime.h"
// $ nvcc -o output example.cu -lcufft

#include <stdio.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define PI 3.14159265358979323846
#define CUDA_CHECK(call) checkCudaError(call, __FILE__, __LINE__)

__device__ void fft(hipDoubleComplex* signal, int n, int offset, int step) {
	if (n <= 1) return;

	hipDoubleComplex even = signal[offset];
	hipDoubleComplex odd = signal[offset + step];

	fft(signal, n / 2, offset, step * 2);
	fft(signal, n / 2, offset + step, step * 2);

	for (int i = 0; i < n / 2; i++) {
		double angle = -2 * PI * i / n;

		hipDoubleComplex temp;
		temp.x = cos(angle) * hipCreal(odd) - sin(angle) * hipCimag(odd);
		temp.y = cos(angle) * hipCimag(odd) + sin(angle) * hipCreal(odd);

		signal[offset + i] = hipCadd(even, temp);
		signal[offset + i + n / 2] = hipCsub(even, temp);

		even = signal[offset + i + step];
	}
}

__global__ void perform_fft(hipDoubleComplex* signal, int n) {
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	fft(signal, n, threadID, 1);
}

void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << "at" << file << ":" << line << std::endl;
        exit(-1);
    }
}

int main()
{       
        // length of the signal
        const int N = 16;
        const int blockSize = 8;

        // Host memory
        hipDoubleComplex *h_signal = new hipDoubleComplex[N];

        // input data initialization
        for (int i = 0; i < N; ++i){
                h_signal[i].x = sin(2 * PI * i / N); // real
                h_signal[i].y = 0; // imaginary
        }

        // Copy to device
        hipDoubleComplex *d_signal;
        CUDA_CHECK(hipMalloc((void**)&d_signal, N * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemcpy(d_signal, h_signal, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        
        // call kernel function
        perform_fft << < (N + blockSize - 1) / blockSize, blockSize >> > (d_signal, N);
        
        // Copy to host
        hipDoubleComplex *h_result = new hipDoubleComplex[N];
        CUDA_CHECK(hipMemcpy(h_result, d_signal, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

        // check the results
        std::cout << "unoptimized" << std::endl;
        for (int i = 0; i< N; ++i){
                std::cout<< "(" << h_result[i].x << ", " << h_result[i].y << ")" << std::endl;
        }

        // deallocate memory
        delete[] h_signal;
        delete[] h_result;
        CUDA_CHECK(hipFree(d_signal));

        return 0;


        // unoptimized
        // (0.382683, 0)
        // (1.44334, -0.146447)
        // (2.55487, -0.541196)
        // (3.42388, -1.20711)
        // (3.77164, -2.07193)
        // (3.39875, -2.97487)
        // (2.23784, -3.69552)
        // (0.382683, -4)
        // (-1.53073, -3.69552)
        // (-1.03153, -2.82843)
        // (0.165911, -2.07193)
        // (1.63099, -1.70711)
        // (2.84776, -1.84776)
        // (3.33809, -2.41421)
        // (2.77904, -3.15432)
        // (-0.382683, 4)
}
